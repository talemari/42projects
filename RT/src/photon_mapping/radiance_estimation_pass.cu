#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   radiance_estimation_pass.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:58 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 21:19:10 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"
#include <hip/hip_runtime.h>

__global__
static void		estimate_region_radiance(t_scene *scene, t_color *pixel_map,
				t_region *region_map, t_tile tile);
__device__
static int		photon_search(t_raytracing_tools *r, t_vec3 *power_added);
__device__
static void		update_region_values(t_raytracing_tools *r, int photons_added,
				t_vec3 power_added);
__device__
static t_vec3	add_accumulated_power(float k, t_region *region,
				t_photon photon, float rad);

void			radiance_estimation_pass(t_raytracing_tools *r, t_tile tile)
{
	dim3 		block_size;
	dim3 		grid_size;
	int			size;

	size = (tile.size / BLOCK_DIM) + ((tile.size % BLOCK_DIM) ? 1 : 0);
	block_size = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	grid_size = dim3(size, size);
	estimate_region_radiance<<<grid_size, block_size>>>(r->d_scene,
		r->d_pixel_map, r->d_region_map, tile);
	cuda_check_kernel_errors();
}

__global__
static void		estimate_region_radiance(t_scene *scene, t_color *pixel_map,
				t_region *region_map, t_tile tile)
{
	t_raytracing_tools	r;
	int					photons_added;
	t_vec3				power_added;

	r.pix.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
	r.idx = scene->res.x * r.pix.y + r.pix.x;
	r.d_region_map = &region_map[(r.pix.x % tile.size) + (r.pix.y % tile.size) *
		tile.size];
	photons_added = 0;
	power_added = v_new(0, 0, 0);
	if (r.pix.x >= scene->res.x || r.pix.y >= scene->res.y ||
		v_isnan(r.d_region_map->hit_pt))
		return ;
	photons_added = photon_search(&r, &power_added);
	update_region_values(&r, photons_added, power_added);
	pixel_map[r.idx] = c_add(pixel_map[r.idx],
		vec_to_col(v_scale(r.d_region_map->power, 1.0 /
	(float)(scene->photon_iteration * scene->photons_per_pass * M_PI *
		r.d_region_map->radius * r.d_region_map->radius))));
}

__device__
static int		photon_search(t_raytracing_tools *r, t_vec3 *power_added)
{
	int 		i;
	float		dist;
	int			photons_added;

	photons_added = 0;
	i = -1;
	while (++i < PHOTON_BOUNCE_MAX * r->scene->photons_per_pass)
	{
		if (!v_isnan(r->scene->photon_list[i].pos))
		{
			dist = v_length(v_sub(r->d_region_map->hit_pt,
				r->scene->photon_list[i].pos));
			if (r->d_region_map->radius > dist && v_dot(r->d_region_map->normal,
				r->scene->photon_list[i].n) > 0.5)
			{
				*power_added = v_add(*power_added,
					add_accumulated_power(r->scene->ppm_light_intensity,
					r->d_region_map, r->scene->photon_list[i], dist * dist));
				++photons_added;
			}
		}
	}
	return (photons_added);
}

__device__
static void		update_region_values(t_raytracing_tools *r, int photons_added,
				t_vec3 power_added)
{
	float	old_rad2;
	float	alpha;

	alpha = 2.0 / 3.0;
	if (photons_added)
	{
		old_rad2 = r->d_region_map->radius * r->d_region_map->radius;
		r->d_region_map->radius *= sqrtf((r->d_region_map->n + alpha *
			photons_added) / (r->d_region_map->n + photons_added));
		r->d_region_map->power = v_scale(v_add(r->d_region_map->power,
			power_added), r->d_region_map->radius * r->d_region_map->radius /
			old_rad2);
	}
	r->d_region_map->n += alpha * photons_added;
}

__device__
static t_vec3	add_accumulated_power(float k, t_region *region,
				t_photon photon, float dist2)
{
	t_vec3 result;

	result = v_scale(col_to_vec(photon.col), sqrtf(region->radius *
		region->radius - dist2) / (M_PI));
	result = v_scale(result, -v_dot(photon.dir, region->normal) * k *
		region->kd);
	return (result);
}
