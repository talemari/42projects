#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cylinder_intersection.cu                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:27:49 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 11:21:17 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
void	get_finite_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
			int index, t_intersection_tools *i);

/*
** Checks for a cylinder-ray intersection.
*/

__device__
bool		get_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
			int index)
{
	t_intersection_tools	i;

	i.v3 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.v1 = v_sub(ray->dir, v_scale(r->scene->objects[index].dir,
		v_dot(ray->dir, r->scene->objects[index].dir)));
	i.q.x = v_dot(i.v1, i.v1);
	i.v2 = v_sub(i.v3, v_scale(r->scene->objects[index].dir,
		v_dot(i.v3, r->scene->objects[index].dir)));
	i.q.y = 2 * v_dot(i.v1, i.v2);
	i.q.z = v_dot(i.v2, i.v2) - pow(r->scene->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	get_finite_cylinder_intersection(r, ray, index, &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CYLINDER;
	}
	return (true);
}

__device__
void	get_finite_cylinder_intersection(t_raytracing_tools *r, t_ray *ray,
			int index, t_intersection_tools *i)
{
	if (i->r1 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r1));
		if (v_dot(r->scene->objects[index].dir,
				v_sub(i->p, r->scene->objects[index].pos)) < 0 ||
			v_dot(r->scene->objects[index].dir,
				v_sub(i->p, v_add(r->scene->objects[index].pos,
				v_scale(r->scene->objects[index].dir,
				r->scene->objects[index].height)))) > 0)
			i->r1 = NAN;
	}
	if (i->r2 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r2));
		if (v_dot(r->scene->objects[index].dir,
				v_sub(i->p, r->scene->objects[index].pos)) < 0 ||
			v_dot(r->scene->objects[index].dir,
				v_sub(i->p, v_add(r->scene->objects[index].pos,
				v_scale(r->scene->objects[index].dir,
				r->scene->objects[index].height)))) > 0)
			i->r2 = NAN;
	}
}

__device__
t_pt2		get_uv_cylinder(t_object *obj, t_ray *ray, t_vec3 *dim)
{
	t_pt2	coord;
	t_vec3	hit_center;

	hit_center = v_scale(ray->nhit, -1);
	coord.x = (0.5 + (atan2f(hit_center.z, hit_center.x) /
		(float)(2 * M_PI))) * dim->x + obj->texture_translate.x;
	coord.y = (sqrtf(v_dist(ray->hit, obj->pos) * v_dist(ray->hit, obj->pos) -
		(obj->rad * obj->rad)) / obj->height) * dim->y +
		obj->texture_translate.y;
	coord.x %= (int)dim->x;
	coord.y %= (int)dim->y;
	return (coord);
}
