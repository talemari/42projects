#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   malloc_region_map.c                                :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 15:14:03 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 16:17:22 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.cuh"

static void init_region_map(t_region *region_map, size_t size,
				float search_rad);

/*
** Allocated or reallocates memory for the CPU region map used in
** Progressive Photon Mapping.
*/

void		malloc_region_map(t_raytracing_tools *r, t_tile tile)
{
	int			i;
	static int	old_tile_size = 0;

	if (!r->scene->is_photon_mapping)
		return ;
	if (r->h_region_map)
	{
		i = -1;
		while (++i < old_tile_size)
			free(r->h_region_map[i]);
		free(r->h_region_map);
		old_tile_size = tile.max;
	}
	if (!(r->h_region_map = (t_region **)malloc(sizeof(t_region *) *
		tile.col * tile.row)))
		exit(0);
	i = -1;
	while (++i < tile.max)
	{
		if (!(r->h_region_map[i] = (t_region *)malloc(sizeof(t_region) *
			tile.size * tile.size)))
			exit(0);
		init_region_map(r->h_region_map[i], tile.size * tile.size,
			r->settings.photon_search_radius);
	}
}

static void	init_region_map(t_region *region_map, size_t size,
				float search_rad)
{
	int i;

	i = -1;
	while (++i < size)
	{
		region_map[i].hit_pt = v_new(NAN, NAN, NAN);
		region_map[i].ray_dir = v_new(NAN, NAN, NAN);
		region_map[i].normal = v_new(NAN, NAN, NAN);
		region_map[i].radius = search_rad;
		region_map[i].n = 0;
		region_map[i].power = v_new(0, 0, 0);
		region_map[i].kd = NAN;
	}
}

/*
** Allocated or reallocates memory for the GPU region map used in
** Progressive Photon Mapping.
*/

void		cuda_malloc_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	size_t	size;

	if (r->scene->is_photon_mapping)
	{
		if (r->d_region_map)
			hipFree(r->d_region_map);
		size = sizeof(t_region) * tile.size * tile.size;
		gpu_errchk(hipMalloc(&(r->d_region_map), size));
	}
}
