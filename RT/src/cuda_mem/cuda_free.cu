/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_free.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 20:50:59 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** Frees lists malloc'ed with hipMalloc.
*/

int	cuda_free(t_raytracing_tools *r, int all)
{
	if ((r->update.resolution == 2 || all) && r->d_pixel_map)
		hipFree(r->d_pixel_map);
	if (r->h_d_scene)
	{
		if ((r->update.objects == 2 || all) && r->h_d_scene->objects)
			hipFree(r->h_d_scene->objects);
		if ((r->update.lights == 2 || all) && r->h_d_scene->lights)
			hipFree(r->h_d_scene->lights);
		if ((r->update.cameras == 2 || all) && r->h_d_scene->cameras)
			hipFree(r->h_d_scene->cameras);
	}
	if ((r->update.scene == 2 || all) && r->d_scene)
		hipFree(r->d_scene);
	return (0);
}
