#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   m_new_rotate_axis.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/26 11:32:07 by tgros             #+#    #+#             */
/*   Updated: 2017/05/26 12:21:14 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Rotates a vector around an arbitrary axis intersecting with the origin
*/

__host__ __device__
void	m_new_rotate_axis(t_matrix *mat, t_vec3 u, float angle)
{
	m_new_identity(mat);
	(*mat)[0][0] = cos(angle) + (u.x * u.x) * (1 - cos(angle));
	(*mat)[0][1] = u.x * u.y * (1 - cos(angle)) - u.z * sin(angle);
	(*mat)[0][2] = u.x * u.z * (1 - cos(angle)) + u.y * sin(angle);
	(*mat)[1][0] = u.y * u.x * (1 - cos(angle)) + u.z * sin(angle);
	(*mat)[1][1] = cos(angle) + u.y * u.y * (1 - cos(angle));
	(*mat)[1][2] = u.y * u.z * (1 - cos(angle)) - u.x * sin(angle);
	(*mat)[2][0] = u.z * u.x * (1 - cos(angle)) - u.y * sin(angle);
	(*mat)[2][1] = u.z * u.y * (1 - cos(angle)) + u.x * sin(angle);
	(*mat)[2][2] = cos(angle) + u.z * u.z * (1 - cos(angle));
}