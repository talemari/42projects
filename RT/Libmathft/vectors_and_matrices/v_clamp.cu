#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   v_clamp.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/22 12:02:26 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:16:43 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../inc/libmathft.cuh"

/*
** Limits the values of the coords of a vec within the range of min and max.
*/

__host__ __device__
t_vec3	v_clamp(t_vec3 v, float min, float max)
{
	v.x < min ? v.x = min : 0;
	v.x > max ? v.x = max : 0;
	v.y < min ? v.y = min : 0;
	v.y > max ? v.y = max : 0;
	v.z < min ? v.z = min : 0;
	v.z > max ? v.z = max : 0;
	return (v);
}
