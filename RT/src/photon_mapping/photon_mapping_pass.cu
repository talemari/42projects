/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_pass.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/29 12:16:47 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 16:58:55 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"
#include <hip/hip_runtime.h>

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count,
				t_photon *init_photon_list);
static float	get_total_intensity(t_light *lights);

void			photon_mapping_pass(t_raytracing_tools *r)
{
	t_photon	*init_photon_list;
	int			photon_count;

	photon_count = r->scene->photons_per_pass;
	gpu_errchk(hipHostMalloc(&init_photon_list, sizeof(t_photon) *
		photon_count));
	init_photon_group(r, photon_count, init_photon_list);
	shoot_photon_wrapper(r, photon_count, init_photon_list);
	hipHostFree(init_photon_list);
}

static void		init_photon_group(t_raytracing_tools *r, size_t photon_count,
				t_photon *init_photon_list)
{
	t_light		*l_ptr;
	float		total_intensity;
	float		ratio;
	int			i;

	total_intensity = get_total_intensity(r->scene->lights);
	l_ptr = r->scene->lights;
	ratio = 0;
	i = -1;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
		{
			l_ptr = l_ptr->next;
			continue ;
		}
		ratio += photon_count * l_ptr->intensity / total_intensity;
		while (++i < ratio)
		{
			init_photon_list[i].pos = l_ptr->pos;
			init_photon_list[i].col = vec_to_col(l_ptr->col);
			init_photon_list[i].n = v_new(NAN, NAN, NAN);
		}
		l_ptr = l_ptr->next;
	}
}

static float	get_total_intensity(t_light *lights)
{
	t_light		*l_ptr;
	float		total_intensity;

	l_ptr = lights;
	total_intensity = 0;
	while (l_ptr)
	{
		if (v_isnan(l_ptr->pos))
		{
			l_ptr = l_ptr->next;
			continue ;
		}
		total_intensity += l_ptr->intensity;
		l_ptr = l_ptr->next;
	}
	return (total_intensity);
}
