#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   init_light_flares.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"

__device__
t_pt2	get_pane_coords(t_ray *ray, t_camera *cam, t_scene *scene);
__global__
void			init_light_flares(t_scene *scene, t_light_flare_tools *tools);
__device__
void		init_ray_and_rt_tools(t_raytracing_tools *t, t_ray *ray,
				t_scene *scene, t_light_flare_tools *tools);
__device__
bool		is_light_blocked(t_raytracing_tools *r, t_ray *ray,
				t_scene *scene, t_light_flare_tools *tools);

void			init_light_flares_wrapper(int light_count,
				t_raytracing_tools *r, t_light_flare_tools *tools)
{
	init_light_flares<<<light_count, 1>>>(r->d_scene, tools);
	cuda_check_kernel_errors();
}

__global__
void			init_light_flares(t_scene *scene, t_light_flare_tools *tools)
{
	t_raytracing_tools	r;
	t_ray				ray;

	r.idx = blockIdx.x;
	init_ray_and_rt_tools(&r, &ray, scene, tools);
	tools[r.idx].is_valid = 1;
	if (!get_view_pane_intersection(&ray, scene->cameras))
	{
		tools[r.idx].is_valid = 0;
		return ;
	}
	tools[r.idx].pos = get_pane_coords(&ray, scene->cameras, scene);
	if (tools[r.idx].pos.x < 0 || tools[r.idx].pos.x >= scene->res.x ||
		tools[r.idx].pos.y < 0 || tools[r.idx].pos.y >= scene->res.y)
	{
		tools[r.idx].is_valid = 0;
		return ;
	}
	tools[r.idx].dist = v_length(v_sub(scene->lights[r.idx].pos, ray.origin));
	tools[r.idx].max_rad = powf(scene->lights[r.idx].intensity, 3) /
		tools[r.idx].dist;
	tools[r.idx].light = &scene->lights[r.idx];
	if (is_light_blocked(&r, &ray, scene, tools))
		tools[r.idx].is_valid = 0;
}

__device__
void		init_ray_and_rt_tools(t_raytracing_tools *r, t_ray *ray,
				t_scene *scene, t_light_flare_tools *tools)
{
	r->scene = scene;
	ray->t = INFINITY;
	ray->type = R_PRIMARY;
	r->t = INFINITY;
	ray->origin = scene->cameras->pos;
	ray->dir = v_norm(v_sub(scene->lights[r->idx].pos, scene->cameras->pos));
}

__device__
bool		is_light_blocked(t_raytracing_tools *r, t_ray *ray,
				t_scene *scene, t_light_flare_tools *tools)
{
	int y;

	if (tools[r->idx].pos.x >= 0 && tools[r->idx].pos.y >= 0 &&
	tools[r->idx].pos.x < scene->res.x && tools[r->idx].pos.y < scene->res.y)
	{
		y = -1;
		while (scene->objects && scene->objects[++y].type != T_INVALID_TOKEN)
		{
			if (intersects(r, ray, y) &&
				ray->t < tools[r->idx].dist && ray->t > 0.0)
				return (true);
		}
	}
	else
		return (true);
	return (false);
}

__device__
t_pt2	get_pane_coords(t_ray *ray, t_camera *cam, t_scene *scene)
{
	t_pt2	coord;
	t_vec3	ortho_x;
	t_vec3	ortho_y;
	t_vec3	proj_dir;

	proj_dir = v_sub(ray->dir, v_scale(cam->dir, -1));
	ortho_x = v_cross(v_new(0, 1, 0), v_scale(cam->dir, -1));
	ortho_y = v_cross(v_scale(cam->dir, -1), ortho_x);
	ortho_x = v_norm(ortho_x);
	ortho_y = v_norm(ortho_y);
	ray->hit = v_add(ray->origin, v_scale(ray->dir, ray->t));
	coord.x = (int)(v_dot(v_scale(ortho_x, -1), proj_dir) * scene->res.x * 0.5 /
		scene->image_aspect_ratio / cam->scale + scene->res.x * 0.5);
	coord.y = (int)(v_dot(v_scale(ortho_y, -1), proj_dir) * scene->res.y * 0.5 /
		cam->scale + scene->res.y / 2);
	return (coord);
}
