/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_objects.cu                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:06:29 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** Allocated memory on the device for objects.
*/

static t_object		*list_to_array_objects(t_object *object);
static size_t		get_objects_array_length(t_object *objects);
static void			update_child_info(t_object *parent, t_object *obj);
static t_object		*list_to_array_objects2(t_object *object, t_object *array);

bool				cuda_malloc_objects(t_raytracing_tools *r, t_scene
					*h_scene_to_array)
{
	if (r->update.objects >= 1)
	{
		h_scene_to_array->objects = list_to_array_objects(r->scene->objects);
		if (r->update.objects == 2)
		{
			if(test_cuda_malloc((void **)(&(r->h_d_scene->objects)), 
				get_objects_array_length(h_scene_to_array->objects)) == false)
				return (false);
		}
		gpu_errchk((hipMemcpy(r->h_d_scene->objects, h_scene_to_array->objects,
			get_objects_array_length(h_scene_to_array->objects),
			hipMemcpyHostToDevice)));
		free(h_scene_to_array->objects);
	}
	return (true);
}

static t_object		*list_to_array_objects(t_object *object)
{
	int			size;
	t_object	*head;
	t_object	*array;

	size = 0;
	head = object;
	while (object)
	{
		++size;
		object = object->next;
	}
	if (!(array = (t_object *)malloc(sizeof(t_object) * (size + 1))))
	{
		write(2, "Malloc error.\n", 14);
		exit(1);
	}
	array[size].type = T_INVALID_TOKEN;
	object = head;
	return (list_to_array_objects2(object, array));
}

static t_object		*list_to_array_objects2(t_object *object, t_object *array)
{
	int size;

	size = -1;
	while (object)
	{
		memcpy(&array[++size], object, sizeof(t_object));
		if (object->parent)
			update_child_info(object->parent, &array[size]);
		object = object->next;
	}
	return (array);
}

static void			update_child_info(t_object *parent, t_object *obj)
{
	obj->pos = v_add(obj->pos, parent->pos);
	obj->pos = p_rotate_axis(v_new(0, 1, 0), parent->dir, parent->pos,
		obj->pos);
	obj->dir = v_norm(p_rotate_axis(v_new(0, 1, 0), parent->dir,
		v_new(0, 0, 0), obj->dir));
	if (parent->parent)
		update_child_info(parent->parent, obj);
}

static size_t		get_objects_array_length(t_object *objects)
{
	size_t	size;

	size = 0;
	if (!objects)
		return (0);
	while (objects[size].type != T_INVALID_TOKEN)
		++size;
	return ((size + 1) * sizeof(t_object));
}
