#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   render.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/01/30 10:59:22 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"

/*
** Calls on the render_pixel kernel to update the d_pixel_map.
*/

__device__
void	render_without_aa(t_raytracing_tools *r);
__device__
void	render_with_aa(t_raytracing_tools *r);
__global__
void	render_pixel(t_scene *scene, t_color *d_pixel_map, t_region *region_map,
		t_tile tile);
void	create_anaglyph_wrapper(t_raytracing_tools *r, dim3 block_size,
		dim3 grid_size, t_tile tile);

void	render(t_raytracing_tools *r, t_tile tile)
{
	dim3			block_size;
	dim3			grid_size;
	int			size;

	size = (tile.size / BLOCK_DIM) + ((tile.size % BLOCK_DIM) ? 1 : 0);
	block_size = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	grid_size = dim3(size, size);
	render_pixel<<<grid_size, block_size>>>(r->d_scene, r->d_pixel_map,
		r->d_region_map, tile);
	cuda_check_kernel_errors();
	if (r->scene->is_3d)
		create_anaglyph_wrapper(r, block_size, grid_size, tile);
}

/*
** Kernel to render a single pixel.
*/

__global__
void	render_pixel(t_scene *scene, t_color *d_pixel_map, t_region *region_map,
		t_tile tile)
{
	t_raytracing_tools	r;

	r.pix.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	r.pix.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
	r.scene = scene;
	r.idx = scene->res.x * r.pix.y + r.pix.x;
	r.d_pixel_map = d_pixel_map;
	if (r.pix.x < scene->res.x && r.pix.y < scene->res.y)
	{
		if (r.scene->is_photon_mapping)
		{
			r.d_region_map = &region_map[(r.pix.x % tile.size) +
				(r.pix.y % tile.size) * tile.size];
		}
		(scene->is_aa == 1 || r.scene->is_photon_mapping) ?
			render_without_aa(&r) :
			render_with_aa(&r);
	}
}

__device__
void	render_without_aa(t_raytracing_tools *r)
{
	t_dpt2	i;
	t_ray	cam_ray;

	i.x = 0.5;
	i.y = 0.5;
	memset(&r->ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
	cam_ray = init_camera_ray(r, i);
	r->d_pixel_map[r->idx] = filter(cast_primary_ray(r, &cam_ray),
		r->scene->cameras->filter);
	__syncthreads();
	update_region_map(r, &cam_ray);
}

__device__
void	render_with_aa(t_raytracing_tools *r)
{
	t_dpt2	aa_i;
	t_ray	cam_ray;
	int		i;
	t_vec3	average;
	float	sample_size;

	sample_size = 1 / (float)r->scene->is_aa;
	aa_i.x = 0.0;
	aa_i.y = 0.0;
	i = -1;
	average = v_new(0, 0, 0);
	while (++i < r->scene->is_aa * r->scene->is_aa)
	{
		aa_i.x += sample_size;
		if (i % (r->scene->is_aa - 1) == 0)
			aa_i.x = 0.0;
		if (i % (r->scene->is_aa - 1) == 0)
			aa_i.y += sample_size;
		memset(&r->ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
		cam_ray = init_camera_ray(r, aa_i);
		average = v_add(average, col_to_vec(cast_primary_ray(r, &cam_ray)));
	}
	average = v_scale(average, 1 / (float)(r->scene->is_aa * r->scene->is_aa));
	r->d_pixel_map[r->idx] = filter(vec_to_col(average),
		r->scene->cameras->filter);
}

