#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_diffuse.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 15:18:12 by talemari          #+#    #+#             */
/*   Updated: 2017/06/03 14:08:19 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the diffuse value of the color at hitpoint.
*/

__device__
t_color	get_diffuse(t_scene *scene, t_ray *primary_ray,
		t_ray *shadow_ray, t_light *light)
{
	t_color	new_col;
	float	r2;

	if (!scene->is_diffuse)
		return (c_new(0, 0, 0));
	r2 = (!v_isnan(light->pos)) ? powf(v_length(v_sub(shadow_ray->origin,
		light->pos)), 2.0) :
		1 / (4 * M_PI);
	new_col = vec_to_col(v_scale(v_mult(light->col,
		v_scale(get_object_color(&scene->objects[primary_ray->hit_obj],
			primary_ray),
		(v_isnan(light->pos) ? light->intensity / 10000 : light->intensity) /
			(4 * M_PI * r2))), ft_clampf(v_dot(shadow_ray->dir,
		v_scale(primary_ray->nhit, primary_ray->n_dir)), 0, 1)));
	new_col = c_scale(new_col, scene->objects[primary_ray->hit_obj].kd);
	return (new_col);
}
