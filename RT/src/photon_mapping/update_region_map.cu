/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   update_region_map.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/28 17:15:06 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 16:57:40 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "hip/hip_runtime.h"
#include "../inc/cuda_call.cuh"

void	copy_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * (tile.col) + (tile.id.x);
		gpu_errchk((hipMemcpy(r->h_region_map[current_tile], r->d_region_map,
			sizeof(t_region) * tile.size * tile.size, hipMemcpyDeviceToHost)));
	}
}

void	get_region_map_tile(t_raytracing_tools *r, t_tile tile)
{
	int current_tile;

	if (r->scene->is_photon_mapping)
	{
		current_tile = (tile.id.y) * (tile.col) + (tile.id.x);
		gpu_errchk((hipMemcpy(r->d_region_map, r->h_region_map[current_tile],
			sizeof(t_region) * tile.size * tile.size, hipMemcpyHostToDevice)));
	}
}

__device__
void	update_region_map(t_raytracing_tools *r, t_ray *cam_ray)
{
	if (r->scene->is_photon_mapping && !v_isnan(cam_ray->hit))
	{
		r->d_region_map->hit_pt = cam_ray->hit;
		r->d_region_map->ray_dir = cam_ray->dir;
		r->d_region_map->normal = v_scale(cam_ray->nhit, cam_ray->n_dir);
		r->d_region_map->kd = r->scene->objects[cam_ray->hit_obj].kd;
	}
}
