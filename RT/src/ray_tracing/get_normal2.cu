#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_normal2.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 16:05:39 by talemari          #+#    #+#             */
/*   Updated: 2017/06/05 15:25:36 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
void		get_cylinder_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;
	float	m;

	x = v_sub(ray->origin, obj->pos);
	m = v_dot(ray->dir, v_scale(obj->dir, ray->t)) + v_dot(x, obj->dir);
	x = v_add(obj->pos, v_scale(obj->dir, m));
	ray->nhit = v_norm(v_sub(ray->hit, x));
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}

__device__
void		get_cone_normal(t_ray *ray, t_object *obj)
{
	t_vec3	x;

	x = v_sub(ray->hit, obj->pos);
	ray->nhit = v_sub(x, v_scale(obj->dir,
		(v_length(x) / cos(obj->angle))));
	ray->nhit = v_norm(ray->nhit);
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}

__device__
void		get_paraboloid_normal(t_ray *ray, t_object *obj)
{
	float	m;

	m = v_dot(v_sub(ray->hit, obj->pos), obj->dir);
	ray->nhit = v_norm(v_sub(v_sub(ray->hit, obj->pos),
		v_scale(obj->dir, obj->height + m)));
	if (obj->normal_map)
		ray->nhit = get_normal_at_normal_map(obj, ray);
}
