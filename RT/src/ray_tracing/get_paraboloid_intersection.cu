#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_paraboloid_intersection.cu                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/16 13:05:58 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 11:21:47 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"

__device__
void	init_paraboloid_roots(t_raytracing_tools *r, t_ray *ray,
			int index, t_intersection_tools *i);

__device__
void		get_finite_paraboloid_intersection(t_raytracing_tools *r, t_ray *ray,
			int index, t_intersection_tools *i)
{
	if (i->r1 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r1));
		if (v_dot(r->scene->objects[index].dir,
			v_sub(i->p, r->scene->objects[index].pos)) < 0 ||
			v_dot(r->scene->objects[index].dir,
			v_sub(i->p, v_add(r->scene->objects[index].pos,
			v_scale(r->scene->objects[index].dir,
			r->scene->objects[index].height)))) > 0)
			i->r1 = NAN;
	}
	if (i->r2 > 0)
	{
		i->p = v_add(ray->origin, v_scale(ray->dir, i->r2));
		if (v_dot(r->scene->objects[index].dir,
			v_sub(i->p, r->scene->objects[index].pos)) < 0 ||
			v_dot(r->scene->objects[index].dir,
			v_sub(i->p, v_add(r->scene->objects[index].pos,
			v_scale(r->scene->objects[index].dir,
			r->scene->objects[index].height)))) > 0)
			i->r2 = NAN;
	}
}

__device__
bool		get_paraboloid_intersection(t_raytracing_tools *r, t_ray *ray,
			int index)
{
	t_intersection_tools	i;

	init_paraboloid_roots(r, ray, index, &i);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	get_finite_paraboloid_intersection(r, ray, index, &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	if (ray->type == R_PRIMARY && r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_CYLINDER;
	}
	return (true);
}

__device__
void	init_paraboloid_roots(t_raytracing_tools *r, t_ray *ray,
			int index, t_intersection_tools *i)
{
	i->v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i->q.x = v_dot(ray->dir, ray->dir) - v_dot(ray->dir,
		r->scene->objects[index].dir) *
		v_dot(ray->dir, r->scene->objects[index].dir);
	i->q.y = 2 * (v_dot(ray->dir, i->v1) - v_dot(ray->dir,
		r->scene->objects[index].dir) *
		(v_dot(i->v1, r->scene->objects[index].dir) + 2 *
		r->scene->objects[index].rad));
	i->q.z = v_dot(i->v1, i->v1) - v_dot(i->v1,
		r->scene->objects[index].dir) * (v_dot(i->v1,
		r->scene->objects[index].dir) + 4 * r->scene->objects[index].rad);
}
