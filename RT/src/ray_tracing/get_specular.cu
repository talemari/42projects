#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_specular.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/11 14:13:51 by talemari          #+#    #+#             */
/*   Updated: 2017/06/04 14:12:41 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the specular value of the color at hitpoint.
*/

__device__
t_color	get_specular(t_scene *scene, t_ray *primary_ray, t_ray *shadow_ray,
		t_light *light)
{
	t_color	new_col;
	float	specular_intensity;
	t_vec3	reflection;
	float	r2;

	if (!scene->is_specular)
		return (c_new(0, 0, 0));
	r2 = v_isnan(light->dir) ? v_dist(primary_ray->hit, light->pos) : 1;
	reflection = reflect(primary_ray->dir, v_scale(primary_ray->nhit,
		primary_ray->n_dir));
	specular_intensity = pow((ft_clampf(v_dot(reflection,
		shadow_ray->dir), 0, 1)),
		scene->objects[primary_ray->hit_obj].specular_exp);
	specular_intensity *= (light->intensity / r2) *
		scene->objects[primary_ray->hit_obj].ks;
	new_col = c_scale(vec_to_col(light->col), specular_intensity);
	return (new_col);
}
