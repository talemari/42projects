#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   in_shadow.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/05 13:13:23 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 11:22:25 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Throws a shadow ray for each light in the scene.
** Returns true if there is an intersection between a light and the origin of
** the ray, else returns false.
*/

__device__
void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k);
__device__
void	filter_color(float *dim, float obj_col, float k);
__device__
void	init_values(t_raytracing_tools *r, float *max, t_light *light,
			t_ray *shadow_ray);

__device__
int			in_shadow(t_raytracing_tools *r, t_ray *shadow_ray,
			t_light *light, t_vec3 *dim_light)
{
	int			i;
	float		max;
	int			is_transparent;

	*dim_light = v_new(1, 1, 1);
	is_transparent = 0;
	init_values(r, &max, light, shadow_ray);
	i = -1;
	while (r->scene->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(r, shadow_ray, i) &&
			shadow_ray->t < max && shadow_ray->t > 0.0)
		{
			shadow_ray->hit = v_add(shadow_ray->origin,
				v_scale(shadow_ray->dir, shadow_ray->t));
			get_normal(shadow_ray, &r->scene->objects[i]);
			if (r->scene->objects[i].transparency < 0.01)
				return (2);
			filter_for_transparency(dim_light,
				get_object_color(&r->scene->objects[i], shadow_ray),
				r->scene->objects[i].transparency);
			is_transparent = 1;
		}
	}
	return (is_transparent);
}

__device__
void	filter_for_transparency(t_vec3 *dim_light, t_vec3 obj_col, float k)
{
	filter_color(&dim_light->x, obj_col.x, k);
	filter_color(&dim_light->y, obj_col.y, k);
	filter_color(&dim_light->z, obj_col.z, k);
}

__device__
void	filter_color(float *dim, float obj_col, float k)
{
	*dim *= (1 - (255 - obj_col) / 255 * (1 - k)) * k;
}

__device__
void	init_values(t_raytracing_tools *r, float *max, t_light *light,
			t_ray *shadow_ray)
{
	r->t = INFINITY;
	*max = (!v_isnan(light->pos)) ?
		v_length(v_sub(light->pos, shadow_ray->origin)) : INFINITY;
}
