#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   update_photon.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/10 15:50:15 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 16:54:41 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__
float			hiprand_uniform (hiprandState_t *state);

__device__
static void		save_photon(t_photon *photon_list, t_ray *ray, t_object *obj,
				t_raytracing_tools *r);
__device__
static float 	get_probability(float *p, t_vec3 o_col, float k, t_vec3 p_col);
__device__
static bool		russian_roullette(t_raytracing_tools *r, t_ray *ray);

__device__
t_color			update_photon(t_raytracing_tools *r, t_ray *ray)
{
	if (r->scene->objects[ray->hit_obj].kd > 0.0 && ray->type ==
		r->scene->is_raytracing ? R_INDIRECT_PHOTON : R_DIRECT_PHOTON)
		save_photon(r->scene->photon_list + r->idx * PHOTON_BOUNCE_MAX,
			ray, &r->scene->objects[ray->hit_obj], r);
	if (russian_roullette(r, ray) &&
		(ray->col.r + ray->col.g + ray->col.b) > 10)
		cast_primary_ray(r, ray);
	return (c_new(0, 0, 0));
}

__device__
static bool		russian_roullette(t_raytracing_tools *r, t_ray *ray)
{
	float	rand_f;
	float	tmp;
	float 	p;

	rand_f = hiprand_uniform(r->devStates);
	tmp = 0;
	p = NAN;
	ray->type = R_INDIRECT_PHOTON;
	if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col,
		r->scene->objects[ray->hit_obj].kd /
		3.0, col_to_vec(ray->col))) > rand_f)
		redirect_photon_diffuse(r, ray, p);
	else if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col,
		r->scene->objects[ray->hit_obj].reflection /
		3.0, col_to_vec(ray->col))) > rand_f)
		redirect_photon_specular(r, ray, p);
	else if ((tmp += get_probability(&p, r->scene->objects[ray->hit_obj].col,
		r->scene->objects[ray->hit_obj].transparency /
		3.0, col_to_vec(ray->col))) > rand_f)
		(fresnel_reflect(r, ray)) ?
			redirect_photon_specular(r, ray, p) :
			redirect_photon_transmit(r, ray, p);
	else
		return (false);
	return (true);
}

__device__
static void		save_photon(t_photon *photon_list, t_ray *ray, t_object *obj,
				t_raytracing_tools *r)
{
	int i;

	i = 0;
	while (!v_isnan(photon_list[i].pos) && i < PHOTON_BOUNCE_MAX && i <
		r->scene->ray_depth)
		++i;
	if (v_isnan(photon_list[i].pos) && i < PHOTON_BOUNCE_MAX && i <
		r->scene->ray_depth)
	{
		photon_list[i].pos = ray->hit;
		photon_list[i].dir = ray->dir;
		photon_list[i].col = ray->type == R_INDIRECT_PHOTON ? ray->col :
			vec_to_col(get_object_color(obj, ray));
		photon_list[i].n = v_scale(ray->nhit, ray->n_dir);
	}
}

__device__
static float	get_probability(float *p, t_vec3 o_col, float k, t_vec3 p_col)
{
	*p = (max(o_col.x * p_col.x, max(o_col.y * p_col.y, o_col.z * p_col.z)) *
		k / 255) / max(p_col.x, max(p_col.y, p_col.z));
	return (*p);
}

__device__
int				fresnel_reflect(t_raytracing_tools *r, t_ray *ray)
{
	float	n1;
	float 	n2;
	float	f;

	if (!r->scene->is_fresnel)
		return (0);
	get_iors(&n1, &n2, r, ray);
	f = get_fresnel_ratio(ray->dir, v_scale(ray->nhit, ray->n_dir), n1, n2);
	return (f > hiprand_uniform_double(r->devStates));
}
