#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cartoon_effect.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/07 20:55:19 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 10:57:31 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../../inc/cuda_call.cuh"

/*
** cartoontools:
** x: bin_size
** y: tolerance
** z: radius_filter
*/

__global__
void	cartoonize(t_scene *scene, t_color *pixel_map, t_vec3 c);

void	get_cartoon_effect(t_raytracing_tools *r)
{
	dim3			block_size;
	dim3			grid_size;
	t_vec3			cartoon_tools;

	cartoon_tools = v_new(45, 50, 2);
	block_size = dim3(32, 32, 1);
	grid_size = dim3(r->scene->res.x / 32 + 1, r->scene->res.y / 32 + 1);
	cartoonize<<<grid_size, block_size>>>(r->d_scene, r->d_pixel_map,
		cartoon_tools);
	cuda_check_kernel_errors();
}

__global__
void	cartoonize(t_scene *scene, t_color *pixel_map, t_vec3 c)
{
	t_pt2	pix;
	int		idx;

	pix.x = blockDim.x * blockIdx.x + threadIdx.x;
	pix.y = blockDim.y * blockIdx.y + threadIdx.y;
	idx = scene->res.x * pix.y + pix.x;
	if (pix.x >= scene->res.x || pix.y >= scene->res.y)
		return ;
	pixel_map[idx].r = pixel_map[idx].r - pixel_map[idx].r % (int)c.x;
	pixel_map[idx].g = pixel_map[idx].g - pixel_map[idx].g % (int)c.x;
	pixel_map[idx].b = pixel_map[idx].b - pixel_map[idx].b % (int)c.x;
	__syncthreads();
}
