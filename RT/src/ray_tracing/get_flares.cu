#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_flares.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 09:56:43 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"

int		get_light_count(t_light *light);
__global__
void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map);

void			get_flares(t_raytracing_tools *r)
{
	int					i;
	t_light_flare_tools *tools;
	size_t				shift;
	int					light_count;

	r->t = INFINITY;
	light_count = get_light_count(r->scene->lights);
	if (!light_count)
		return ;
	if (!test_cuda_malloc((void **)&tools, sizeof(t_light_flare_tools) * light_count))
		exit (1);
	init_light_flares_wrapper(light_count, r, tools);
	i = -1;
	shift = 0;
	while (++i < light_count)
	{
		dim3 block_size = dim3(BLOCK_DIM, BLOCK_DIM, 1);
		dim3 grid_size = dim3(r->scene->res.x / BLOCK_DIM + 1,
			r->scene->res.y / BLOCK_DIM + 1);
		draw_one_flare<<<grid_size, block_size>>>(tools + shift,
			r->d_scene, r->d_pixel_map);
		cuda_check_kernel_errors();
		shift++;
	}
}

int		get_light_count(t_light *light)
{
	int				i;
	t_light			*l_ptr;

	i = 0;
	l_ptr = light;
	while (l_ptr)
	{
		++i;
		l_ptr = l_ptr->next;
	}
	return (i);
}

__global__
void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map)
{
	t_pt2	pix;
	float	rad;
	t_color	col;

	pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (pix.x < tools->pos.x - tools->max_rad ||
		pix.x > tools->pos.x + tools->max_rad ||
		pix.y < tools->pos.y - tools->max_rad ||
		pix.y > tools->pos.y + tools->max_rad ||
		pix.x < 0 || pix.x >= scene->res.x || pix.y < 0 ||
		pix.y >= scene->res.y || !tools->is_valid)
		return ;
	rad = sqrt(powf((tools->pos.x - pix.x), 2) +
		powf((tools->pos.y - pix.y), 2));
	col = pixel_map[pix.y * scene->res.x + pix.x];
	col = c_add(col, c_scale(vec_to_col(tools->light->col),
		((tools->max_rad - rad) /
		(tools->max_rad * rad)) * tools->light->kflare));
	pixel_map[pix.y * scene->res.x + pix.x] = col;
}
