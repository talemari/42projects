/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_camera.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:13:20 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** Allocated memory on the device for the first camera.
*/

bool		cuda_malloc_camera(t_raytracing_tools *r)
{
	if (r->update.cameras >= 1)
	{
		if (r->update.cameras == 2)
		{
			if(test_cuda_malloc((void **)(&r->h_d_scene->cameras), sizeof(t_camera)) == false)
				return(false);
		}
		if (r->scene->is_3d)
			r->scene->cameras->filter = F_LEFT_RED;
		gpu_errchk((hipMemcpy(r->h_d_scene->cameras, r->scene->cameras,
			sizeof(t_camera), hipMemcpyHostToDevice)));
	}
	return(true);
}
