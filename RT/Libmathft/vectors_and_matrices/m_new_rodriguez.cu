#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   m_new_rodriguez.cu                                 :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:18:18 by jwalsh            #+#    #+#             */
/*   Updated: 2017/06/09 09:31:24 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Calculates and returns the roation matrix from vector b to vector a.
*/

__host__ __device__
void	get_value_rodriguez(t_matrix n, t_vec3 v)
{
	n[1][0] = v.z;
	n[2][0] = -v.y;
	n[0][1] = -v.z;
	n[2][1] = v.x;
	n[0][2] = v.y;
	n[1][2] = -v.x;
}

__host__ __device__
void	m_new_rodriguez(t_matrix *m, t_vec3 a, t_vec3 b)
{
	t_matrix	n;
	t_matrix	identity;
	t_vec3		v;
	double		angle;
	int			i;

	v = v_norm(v_cross(a, b));
	angle = acos(v_dot(a, b) / ((v_length(a) * v_length(b))));
	m_new_identity(&n);
	m_new_identity(&identity);
	get_value_rodriguez(n, v);
	m_scale(&n, sin(angle));
	m_add(&n, &identity);
	memcpy(m, &n, sizeof(t_matrix));
	m_mult(&n, &n);
	m_scale(&n, 1 - cos(angle));
	m_add(m, &n);
	i = -1;
	while (++i < 3)
	{
		(*m)[3][i] = 0;
		(*m)[i][3] = 0;
	}
	(*m)[3][3] = 1;
}
