#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_clampf.cu                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/22 14:13:21 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:28:43 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

/*
** Changes the value of int i to max if i > max, or to min if i < min.
*/

#include "libmathft.cuh"

__host__ __device__
float	ft_clampf(float i, float min, float max)
{
	i < min ? i = min : 0;
	i > max ? i = max : 0;
	return (i);
}
