#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   p_rotate_axis.cu                                   :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/26 11:38:22 by tgros             #+#    #+#             */
/*   Updated: 2017/05/26 12:11:19 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Rotates a point around an arbitrary axis intersecting an arbitrary point
*/

__host__ __device__
t_vec3		p_rotate_axis(t_vec3 d1, t_vec3 d2, t_vec3 rot_p, t_vec3 point)
{
	t_matrix	mat;
	t_vec3		res;
	t_vec3		u;
	float		angle;

	d1 = v_norm(d1);
	d2 = v_norm(d2);
	u = v_norm(v_cross(d2, d1));
	angle = acos((v_dot(d1, d2)) / (v_length(d1) * v_length(d2)));
	m_new_rotate_axis(&mat, u, angle);

	res = m_v_mult(v_sub(point, rot_p), &mat);
	res = v_add(res, rot_p);
	return (res);
}