/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_check_kernel_errors.cu                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/03 14:13:47 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 20:32:50 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** An error checking function to be used agfter every kernel launch.
*/

void	cuda_check_kernel_errors(void)
{
	hipError_t	errSync;
	hipError_t	errAsync;
	const char	*s;
	int			l;

	errSync = hipGetLastError();
	errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
	{
		write(2, "\e[1;93mSync kernel error: ", 26);
		s =  hipGetErrorString(errSync);
		l = strlen(s);
		write(2, s, l);
		write(2, "\e[0m\n", 5);
	}
	if (errAsync != hipSuccess)
	{
		write(2, "\e[1;93mAsync kernel error: ", 27);
		s =  hipGetErrorString(errAsync);
		l = strlen(s);
		write(2, s, l);
		write(2, "\e[0m\n", 5);
	}
	if (errSync != hipSuccess || errAsync != hipSuccess)
		exit(-1);
}
