#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   v_isnan.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:25:01 by jwalsh            #+#    #+#             */
/*   Updated: 2017/03/31 15:26:14 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Checks if a vector's coordinates are numbers.
** Returns 1 if all coords are NAN.
*/

__host__ __device__
int	v_isnan(t_vec3 v)
{
	return (isnan(v.x) && isnan(v.y) && isnan(v.z));
}
