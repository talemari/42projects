/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   redirect_photon.cu                                 :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 16:00:41 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 16:45:00 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__
static void		scale_color(t_ray *ray, float p, float k, t_vec3 obj_col);

/*
** Redirection functions based on Russian Roullette result.
*/

__device__
void			redirect_photon_diffuse(t_raytracing_tools *r, t_ray *ray,
				float p)
{
	t_vec3			rand_dir;
	hiprandState		state;

	ray->origin = v_add(ray->hit, v_scale(ray->nhit, ray->n_dir * BIAS));
	state = *r->devStates;
	rand_dir.x = r->rand_list[(r->idx + 3) % (r->scene->photons_per_pass * 3)];
	rand_dir.y = r->rand_list[(r->idx + 4) % (r->scene->photons_per_pass * 3)];
	rand_dir.z = r->rand_list[(r->idx + 5) % (r->scene->photons_per_pass * 3)];
	*r->devStates = state;
	rand_dir = v_norm(rand_dir);
	ray->dir = (v_dot(rand_dir, v_scale(ray->nhit, ray->n_dir)) < 0) ?
		v_scale(rand_dir, -1) : rand_dir;
	scale_color(ray, p, r->scene->objects[ray->hit_obj].kd,
		r->scene->objects[ray->hit_obj].col);
}

__device__
void			redirect_photon_specular(t_raytracing_tools *r, t_ray *ray,
				float p)
{
	ray->origin = v_add(ray->hit, v_scale(ray->nhit, ray->n_dir * BIAS));
	ray->dir = reflect(ray->dir, v_scale(ray->nhit, ray->n_dir));
	scale_color(ray, p, r->scene->objects[ray->hit_obj].reflection,
		r->scene->objects[ray->hit_obj].col);
}

__device__
void			redirect_photon_transmit(t_raytracing_tools *r, t_ray *ray,
				float p)
{
	float n1;
	float n2;

	update_ior(&n1, &n2, r, ray);
	ray->ior = n2;
	ray->origin = v_add(ray->hit, v_scale(ray->nhit, -ray->n_dir * BIAS));
	ray->dir = refract(ray->dir, v_scale(ray->nhit, ray->n_dir), n1, n2);
	scale_color(ray, p, r->scene->objects[ray->hit_obj].transparency,
		r->scene->objects[ray->hit_obj].col);
}

/*
** ray: incoming ray.
** p: probability of particular redirection (diffuse, refelcted, refracted)
** k: coefficient of redirection (diffuse, reflection, transparency)
** obj_col: color of object hit
** P_refl = P_inc * k / p
*/

__device__
static void		scale_color(t_ray *ray, float p, float k, t_vec3 obj_col)
{
	t_color p_inc;

	p_inc = ray->col;
	ray->col.r = p_inc.r * k * (obj_col.x / (p * 255.0));
	ray->col.g = p_inc.g * k * (obj_col.y / (p * 255.0));
	ray->col.b = p_inc.b * k * (obj_col.z / (p * 255.0));
}
