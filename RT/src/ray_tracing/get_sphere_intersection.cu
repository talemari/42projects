#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_sphere_intersection.cu                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:26:41 by talemari          #+#    #+#             */
/*   Updated: 2017/06/04 14:12:42 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Checks for a ray-sphere intersection.
*/

__device__
bool	get_sphere_intersection(t_raytracing_tools *r, t_ray *ray, int index)
{
	t_intersection_tools	i;

	i.v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir);
	i.q.y = 2 * v_dot(i.v1, ray->dir);
	i.q.z = v_dot(i.v1, i.v1) - pow(r->scene->objects[index].rad, 2);
	if (!solve_quadratic(i.q, &i.r1, &i.r2))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	(i.r1 < 0) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0)
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0.0;
	if (r->t > i.r1)
	{
		ray->hit_obj = index;
		ray->hit_type = T_SPHERE;
	}
	return (true);
}

__device__
t_pt2	get_uv_sphere(t_object *obj, t_ray *ray, t_vec3 *dim)
{
	t_pt2	coord;
	t_vec3	hit_center;

	hit_center = v_scale(ray->nhit, -1);
	coord.x = (0.5 + (atan2f(hit_center.z, hit_center.x) /
		(float)(2 * M_PI))) * dim->x + obj->texture_translate.x;
	coord.y = (0.5 - (asinf(hit_center.y) / M_PI)) * dim->y +
		obj->texture_translate.y;
	coord.x %= (int)obj->texture_dim.x;
	coord.y %= (int)obj->texture_dim.y;
	return (coord);
}
