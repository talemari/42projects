#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   refract.cu                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 15:37:56 by talemari          #+#    #+#             */
/*   Updated: 2017/06/04 15:25:27 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Given a ray direction and normal at hit point, returns the direction of the
** refracted ray.
*/

__device__
t_vec3	refract(t_vec3 ray_dir, t_vec3 nhit, float ray_ior, float new_ior)
{
	float	cosi;
	t_vec3	eta;
	float	k;
	t_vec3	n;

	cosi = ft_clampf(v_dot(ray_dir, nhit), -1, 1);
	eta.x = ray_ior;
	eta.y = new_ior;
	n = nhit;
	if (cosi < 0)
		cosi = -cosi;
	else
	{
		ft_swapf(&eta.x, &eta.y);
		n = v_scale(nhit, -1);
	}
	eta.z = eta.x / eta.y;
	k = 1 - eta.z * eta.z * (1 - cosi * cosi);
	return (k < 0 ? v_new(NAN, NAN, NAN) : v_add(v_scale(ray_dir, eta.z),
		v_scale(n, (eta.z * cosi - sqrtf(k)))));
}
