#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_fresnel_ratio.cu                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 14:25:09 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 11:23:53 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the fresnel ratio.
*/

__device__
float	get_fresnel_ratio2(float cosi, float etai,
				float etat, float sint);

__device__
float			get_fresnel_ratio(t_vec3 ray_dir, t_vec3 normal,
				float n1, float n2)
{
	float	cosi;
	float	etai;
	float	etat;
	float	sint;

	cosi = ft_clampf(v_dot(ray_dir, normal), -1, 1);
	etai = n1;
	etat = n2;
	if (cosi > 0)
		ft_swapf(&etai, &etat);
	sint = (etai / etat) * (1 - cosi * cosi > 0.0 ?
		sqrtf(1 - cosi * cosi) : 0.0);
	if (sint >= 1)
		return (1);
	else
		return (get_fresnel_ratio2(cosi, etai, etat, sint));
}

__device__
float	get_fresnel_ratio2(float cosi, float etai,
				float etat, float sint)
{
	float	cost;
	float	rs;
	float	rp;

	cost = sqrtf((1 - sint * sint > 0 ? 1 - sint * sint : 0.0));
    cosi = cosi < 0 ? -cosi : cosi;
    rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
    rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
	return ((rs * rs + rp * rp) / 2);
}
