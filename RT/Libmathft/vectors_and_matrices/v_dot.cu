#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   v_dot.cu                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:24:27 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:16:55 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** The dot product of two vectors relates to the cosine of the angle between
** the two vectors.
**
** If B is a unit vector, then the product A.B gives ||A||cos(a), The
** magnitude of the projection of A in teh direction of B, with a minus
** sign if the direction is opposite.
**
** When neither A nor B is a unit vector, get the unit vector of one of The
** vectors: A.B/||B||.
**
** When both vectors are normalized, taking the arc cosine of the dot product
** gives you the angle a between the two vectors. (acos(x), x in radians)
*/

__host__ __device__
float	v_dot(t_vec3 vec1, t_vec3 vec2)
{
	return (vec1.x * vec2.x +
			vec1.y * vec2.y +
			vec1.z * vec2.z);
}
