#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   shoot_photon.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 14:38:56 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 09:57:41 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
** Kernel wrapper and kernel for shooting photon
*/

static void		init_random_numbers(int nb, float *random_numbers);
__global__
static void		shoot_photon(t_scene *scene, t_photon *init_photon_list,
				int photon_count, float *rand_numbers);
__device__
static t_ray	init_kernel_photon(t_raytracing_tools *r, t_photon photon,
				float *rand_numbers);
__device__
static void		set_default_photon_values(t_photon *p_list, int idx, int depth);

void			shoot_photon_wrapper(t_raytracing_tools *r, size_t photon_count,
				t_photon *init_photon_list)
{
	dim3		block_size;
	dim3		grid_size;
	float		*h_rand_numbers;
	float		*d_rand_numbers;
	int			rand_size;

	rand_size = r->scene->photons_per_pass * 3;
	block_size = dim3(BLOCK_DIM, 1, 1);
	grid_size = dim3(photon_count / BLOCK_DIM + 1, 1);
	if (!(h_rand_numbers = (float *)malloc(sizeof(float) * rand_size)))
		exit (1);
	if (!test_cuda_malloc((void **)&d_rand_numbers, sizeof(float) * rand_size))
		exit (1);
	init_random_numbers(rand_size, h_rand_numbers);
	hipMemcpy(d_rand_numbers, h_rand_numbers, sizeof(float) * rand_size,
		hipMemcpyHostToDevice);
	shoot_photon<<<grid_size, block_size>>>(r->d_scene, init_photon_list,
		photon_count, d_rand_numbers);
	cuda_check_kernel_errors();
	hipFree(d_rand_numbers);
	free(h_rand_numbers);
}

/*
** Kernel for shooting a single photon
*/

__global__
static void		shoot_photon(t_scene *scene, t_photon *init_photon_list,
				int photon_count, float *rand_numbers)
{
	t_raytracing_tools	r;
	t_ray				photon;
	hiprandState			state;

	r.pix.x = 0;
	r.pix.y = 0;
	r.scene = scene;
	r.idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	r.rand_list = rand_numbers;
	if (r.idx >= photon_count)
		return ;
	r.devStates = &state;
	hiprand_init(rand_numbers[0], r.idx % (int)rand_numbers[0], r.idx *
		rand_numbers[0], r.devStates);
	memset(&r.ior_list, 0, sizeof(float) * (PHOTON_BOUNCE_MAX + 1));
	photon = init_kernel_photon(&r, init_photon_list[r.idx], rand_numbers);
	if (photon.col.r + photon.col.g + photon.col.b > 20)
		cast_primary_ray(&r, &photon);
}

__device__
static t_ray	init_kernel_photon(t_raytracing_tools *r, t_photon photon,
				float *rand_numbers)
{
	t_ray		new_ray;

	new_ray.dir.x = rand_numbers[r->idx];
	new_ray.dir.y = rand_numbers[r->idx + 1];
	new_ray.dir.z = rand_numbers[r->idx + 2];
	new_ray.dir = v_norm(new_ray.dir);
	new_ray.type = R_DIRECT_PHOTON;
	new_ray.origin = photon.pos;
	new_ray.col = photon.col;
	new_ray.nhit = photon.n;
	new_ray.n_dir = 1;
	new_ray.depth = r->scene->ray_depth;
	new_ray.ior = r->scene->cameras->ior;
	r->ior_list[0] = r->scene->cameras->ior;
	set_default_photon_values(r->scene->photon_list, r->idx,
		r->scene->ray_depth);
	return (new_ray);
}

__device__
static void		set_default_photon_values(t_photon *p_list, int idx, int depth)
{
	int i;

	i = 0;
	while (i < depth && i < PHOTON_BOUNCE_MAX)
	{
		p_list[idx * PHOTON_BOUNCE_MAX + i].pos = v_new(NAN, NAN, NAN);
		p_list[idx * PHOTON_BOUNCE_MAX + i].dir = v_new(NAN, NAN, NAN);
		p_list[idx * PHOTON_BOUNCE_MAX + i].n = v_new(NAN, NAN, NAN);
		p_list[idx * PHOTON_BOUNCE_MAX + i].col = c_new(0, 0, 0);
		++i;
	}
}

static void		init_random_numbers(int nb, float *random_numbers)
{
	int i;

	i = -1;
	while (++i < nb)
		random_numbers[i] = (((float)rand()) / (float)INT_MAX) - 0.5;
}
