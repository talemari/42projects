/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc_lights.cu                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/18 16:10:25 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 09:31:43 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

/*
** Allocated memory on the device for lights.
*/

static t_light		*list_to_array_lights(t_light *light);
static size_t		get_lights_array_length(t_light *lights);

bool				cuda_malloc_lights(t_raytracing_tools *r,
					t_scene *h_scene_to_array)
{
	if (r->update.lights >= 1)
	{
		h_scene_to_array->lights = list_to_array_lights(r->scene->lights);
		if (r->update.lights == 2)
		{
			if(test_cuda_malloc((void **)(&(r->h_d_scene->lights)), 
				get_lights_array_length(h_scene_to_array->lights)) == false)
				return(false);
		}
		gpu_errchk((hipMemcpy(r->h_d_scene->lights, h_scene_to_array->lights,
			get_lights_array_length(h_scene_to_array->lights),
			hipMemcpyHostToDevice)));
		free(h_scene_to_array->lights);
	}
	return(true);
}

static t_light		*list_to_array_lights(t_light *light)
{
	int			size;
	t_light		*head;
	t_light		*array;

	size = 0;
	head = light;
	while (light)
	{
		++size;
		light = light->next;
	}
	if (!(array = (t_light *)malloc(sizeof(t_light) * (size + 1))))
		write(2, "Malloc error.\n", 14);
	if (!array)
		exit(1);
	array[size].col = v_new(NAN, NAN, NAN);
	light = head;
	size = -1;
	while (light)
	{
		array[++size].col = v_new(NAN, NAN, NAN);
		memcpy(&array[size], light, sizeof(t_light));
		light = light->next;
	}
	return (array);
}

static size_t		get_lights_array_length(t_light *lights)
{
	size_t	size;

	size = 0;
	while (!v_isnan(lights[size].col))
		++size;
	return ((size + 1) * sizeof(t_light));
}
