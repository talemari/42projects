#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   m_new_rotate.cu                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:22:02 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 13:01:03 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

__host__ __device__
static void m_rot_x(t_matrix *m, float angle);
__host__ __device__
static void m_rot_y(t_matrix *m, float angle);
__host__ __device__
static void m_rot_z(t_matrix *m, float angle);

/*
** Creates a new rotation matrix of angle 'angle' around axis 'axis'
*/

__host__ __device__
void		m_new_rotate(t_matrix *m, float angle, char axis)
{
	if (axis == 'x' || axis == 'X')
		m_rot_x(m, angle);
	else if (axis == 'y' || axis == 'Y')
		m_rot_y(m, angle);
	else if (axis == 'z' || axis == 'Z')
		m_rot_z(m, angle);
}

__host__ __device__
static void m_rot_x(t_matrix *m, float angle)
{
	m_new(m);
	(*m)[0][0] = 1;
	(*m)[1][1] = cos(to_radian(angle));
	(*m)[1][2] = sin(to_radian(angle));
	(*m)[2][1] = -sin(to_radian(angle));
	(*m)[2][2] = cos(to_radian(angle));
}

__host__ __device__
static void m_rot_y(t_matrix *m, float angle)
{
	(*m)[0][0] = cos(to_radian(angle));
	(*m)[0][2] = -sin(to_radian(angle));
	(*m)[1][1] = 1;
	(*m)[2][0] = sin(to_radian(angle));
	(*m)[2][2] = cos(to_radian(angle));
}

__host__ __device__
static void m_rot_z(t_matrix *m, float angle)
{
	m_new(m);
	(*m)[0][0] = cos(to_radian(angle));
	(*m)[0][1] = sin(to_radian(angle));
	(*m)[1][0] = -sin(to_radian(angle));
	(*m)[1][1] = cos(to_radian(angle));
	(*m)[2][2] = 1;
}
