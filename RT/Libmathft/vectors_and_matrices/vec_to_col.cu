#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   vec_to_col.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/19 13:29:54 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/19 13:34:24 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Converts a t_vec3 to a t_color
*/

__host__ __device__
t_color		vec_to_col(t_vec3 vec)
{
	t_color	col;

	if (vec.x > 255)
		col.r = 255;
	else if (vec.x < 0 || isnan(vec.x))
		col.r = 0;
	else
		col.r = (uint8_t)vec.x;
	if (vec.y > 255)
		col.g = 255;
	else if (vec.y < 0 || isnan(vec.y))
		col.g = 0;
	else
		col.g = (uint8_t)vec.y;
	if (vec.z > 255)
		col.b = 255;
	else if (vec.z < 0 || isnan(vec.z))
		col.b = 0;
	else
		col.b = (uint8_t)vec.z;
	return (col);
}