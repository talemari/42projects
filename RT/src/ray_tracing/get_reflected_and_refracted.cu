#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_reflected_and_refracted.cu                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/15 13:49:42 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 15:50:44 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

/*
** Calculates and returns the reflected value of the color at hitpoint.
*/

__device__
t_color	get_refracted(t_raytracing_tools *r, t_scene *scene,
				t_ray *ray);
__device__
t_color	get_reflected(t_raytracing_tools *r, t_scene *scene,
				t_ray *ray, float f);
__device__
t_color	get_beer_lambert_color(t_raytracing_tools *r, t_ray *ray,
				t_color col, float kt);

__device__
t_color			get_reflected_and_refracted(t_raytracing_tools *r,
				t_scene *scene, t_ray *ray)
{
	if (scene->objects[ray->hit_obj].transparency > 0.0001)
		return (get_refracted(r, scene, ray));
	else if (scene->objects[ray->hit_obj].reflection > 0.0001)
		return (get_reflected(r, scene, ray, 0));
	else
		return (c_new(0, 0, 0));
}

__device__
t_color	get_refracted(t_raytracing_tools *r, t_scene *scene,
				t_ray *ray)
{
	t_ray		refr;
	float		n1;
	float		n2;
	float		f;
	t_object	*o;

	update_ior(&n1, &n2, r, ray);
	o = &scene->objects[ray->hit_obj];
	refr.t = INFINITY;
	refr.type = R_PRIMARY;
	refr.depth = ray->depth;
	refr.ior = n2;
	refr.origin = v_add(ray->hit, v_scale(ray->nhit, -ray->n_dir * BIAS));
	refr.dir = refract(ray->dir, v_scale(ray->nhit, ray->n_dir), n1, n2);
	if (v_isnan(refr.dir))
		return (get_reflected(r, scene, ray, o->transparency - o->reflection));
	f = scene->is_fresnel ? get_fresnel_ratio(ray->dir,
		v_scale(ray->nhit, ray->n_dir), n1, n2) : 0;
	if (scene->is_fresnel || o->reflection > 0.0)
		return (c_add(c_scale(get_beer_lambert_color(r, &refr,
		cast_primary_ray(r, &refr), o->transparency), (1 - f) *
		o->transparency), get_reflected(r, scene, ray, f)));
	return (c_scale(get_beer_lambert_color(r, &refr,
		cast_primary_ray(r, &refr), o->transparency), o->transparency));
}

/*
** Changes color based on Beer Lambert formula, reducing transmitted color based
** on coefficient and distance.
*/

__device__
t_color	get_beer_lambert_color(t_raytracing_tools *r, t_ray *ray,
				t_color col, float kt)
{
	if (ray->t != INFINITY)
		return (c_scale(col, exp((r->scene->objects[ray->hit_obj].beer_lambert) *
		ray->t * (1.0 - kt))));
	return (col);
}

__device__
t_color	get_reflected(t_raytracing_tools *r, t_scene *scene,
				t_ray *ray, float f)
{
	t_ray	reflected;

	reflected.type = R_PRIMARY;
	reflected.origin = v_add(ray->hit, v_scale(ray->nhit, ray->n_dir * BIAS));
	reflected.dir = reflect(ray->dir, v_scale(ray->nhit, ray->n_dir));
	reflected.ior = ray->ior;
	reflected.depth = ray->depth;
	reflected.n_dir = 1;
	reflected.t = INFINITY;
	return (c_scale(cast_primary_ray(r, &reflected), f +
		scene->objects[ray->hit_obj].reflection));
}
