#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   draw_one_flare.cu                                  :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/23 10:18:02 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 14:14:09 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/cuda_call.cuh"
#include "rt.cuh"
#include "hip/hip_runtime.h"

__global__
void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map);

void	draw_one_flare_wrapper(t_raytracing_tools *r, t_light_flare_tools *tools, int light_count)
{
	int i;
	size_t shift;
	dim3 grid_size;
	dim3 block_size;

	i = -1;
	shift = 0;
	while (++i < light_count)
	{
		block_size = dim3(BLOCK_DIM, BLOCK_DIM, 1);
		grid_size = dim3(r->scene->res.x / BLOCK_DIM + 1,
			r->scene->res.y / BLOCK_DIM + 1);
		draw_one_flare<<<grid_size, block_size>>>(tools + shift,
			r->d_scene, r->d_pixel_map);
		cuda_check_kernel_errors();
		shift++;
	}
}

__global__
void		draw_one_flare(t_light_flare_tools *tools, t_scene *scene,
				t_color *pixel_map)
{
	t_pt2	pix;
	float	rad;
	t_color	col;

	pix.x = (blockDim.x * blockIdx.x) + threadIdx.x;
	pix.y = (blockDim.y * blockIdx.y) + threadIdx.y;
	if (pix.x < tools->pos.x - tools->max_rad ||
		pix.x > tools->pos.x + tools->max_rad ||
		pix.y < tools->pos.y - tools->max_rad ||
		pix.y > tools->pos.y + tools->max_rad ||
		pix.x < 0 || pix.x >= scene->res.x || pix.y < 0 ||
		pix.y >= scene->res.y || !tools->is_valid)
		return ;
	rad = sqrt(powf((tools->pos.x - pix.x), 2) +
		powf((tools->pos.y - pix.y), 2));
	col = pixel_map[pix.y * scene->res.x + pix.x];
	col = c_add(col, c_scale(vec_to_col(tools->light->col),
		((tools->max_rad - rad) /
		(tools->max_rad * rad)) * tools->light->kflare));
	pixel_map[pix.y * scene->res.x + pix.x] = col;
}