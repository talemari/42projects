/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   test_cuda_malloc.cu                                :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 14:56:48 by talemari          #+#    #+#             */
/*   Updated: 2017/06/07 14:57:03 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include <stdlib.h>

/*
** Tries to call hipMalloc until success. Displays popup after 5 tries.
*/

void	malloc_error(char *cuda, const char *s)
{
	write(2, "\e[1;93mcudaMalloc fail after ", 29);
	write(2, cuda, 1);
	write(2, " attempts\n", 10);
	write(2, "GPUassert : ", 12);
	write(2, s, strlen(s));
	write(2, "\e[0m\n", 5);
}

bool	test_cuda_malloc(void **to_malloc, size_t size)
{
	int i;
	const char *s;
	char cuda[1];
	int code;
	int test_cuda;

	i = 0;
	test_cuda = CUDA_TEST;
	if (test_cuda < 1 && test_cuda > 9)
		test_cuda = 5;
	while (i < test_cuda)
	{
		if ((code = hipMalloc(to_malloc, size)) == 0)
			return (true);
		i++;
	}
	s = hipGetErrorString((hipError_t)code);
	cuda[0] = test_cuda + 48;
	if (code != hipSuccess)
		malloc_error(cuda, s);
	return (false);
}
