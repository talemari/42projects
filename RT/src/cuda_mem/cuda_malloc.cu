/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cuda_malloc.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/22 12:51:28 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 15:51:14 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"
#include "../inc/cuda_call.cuh"

static void		reset_update_struct(t_raytracing_tools *r);
bool			cuda_malloc_scene(t_raytracing_tools *r);

/*
** Allocates memory on the device and on pinned memory the various sturctures
** in the scene.
*/

bool				cuda_malloc(t_raytracing_tools *r)
{
	t_scene		h_scene_to_array;

	memcpy(&h_scene_to_array, r->scene, sizeof(t_scene) - (sizeof(void *) * 3));
	memcpy(r->h_d_scene, r->scene, sizeof(t_scene) - (sizeof(void *) * 3));
	if (cuda_malloc_objects(r, &h_scene_to_array) == false)
		return(false);
	if (cuda_malloc_lights(r, &h_scene_to_array) == false)
		return(false);
	if (cuda_malloc_camera(r) == false)
		return(false);
	if (cuda_malloc_scene(r) == false)
		return(false);
	gpu_errchk(hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice));
	reset_update_struct(r);
	return (true);
}

static void		reset_update_struct(t_raytracing_tools *r)
{
	r->update.resolution = 0;
	r->update.objects = 0;
	r->update.lights = 0;
	r->update.cameras = 0;
	r->update.scene = 0;
	r->update.ray_depth = 0;
	r->update.render = 0;
	r->update.photon_map = 0;
}

 bool		cuda_malloc_scene(t_raytracing_tools *r)
{
	if (r->update.resolution == 2)
	{
		gpu_errchk((hipHostMalloc(&r->d_pixel_map, sizeof(t_color) *
			r->scene->res.y * r->scene->res.x)));
		if (r->scene->is_3d)
			gpu_errchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) *
				r->scene->res.y * r->scene->res.x)));
	}
	if (r->update.ray_depth == 2)
	{
		gpu_errchk(hipSetDevice(0));
		hipDeviceSetLimit(hipLimitStackSize, 1024 * MAX_RAY_DEPTH);
	}
	if (r->update.anaglyph == 2)
		gpu_errchk((hipHostMalloc(&r->d_pixel_map_3d, sizeof(t_color) *
			r->scene->res.y * r->scene->res.x)));
	if (r->update.scene == 2)
	{
		if(test_cuda_malloc((void **)(&r->d_scene), sizeof(t_scene)) == false)
			return(false);
	}
	return(true);
}
