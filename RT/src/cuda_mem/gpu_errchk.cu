/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   gpu_errchk.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/04 15:30:53 by talemari          #+#    #+#             */
/*   Updated: 2017/06/05 10:57:14 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "rt.cuh"

/*
** Checks a cuda function return value for errors and exits with an error 
** message.
*/

__host__
void gpu_errchk(int code)
{
	int			l;
	const char	*s;

	s = hipGetErrorString((hipError_t)code);
	l = strlen(s);
	if (code != hipSuccess)
	{
		write(2, "\e[1;93mGPUassert: ", 17);
		write(1, "test\n", 5);
		write(2, s, l);
		write(2, "\e[0m\n", 5);
		exit(code);
	}
}
