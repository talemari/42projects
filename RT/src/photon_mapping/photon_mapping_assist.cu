/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   photon_mapping_assist.cu                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/08 12:55:39 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 10:18:55 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "cuda_call.cuh"
#include <hip/hip_runtime.h>

/*
** Functions assiting PPM.
*/

void	perpare_memory(t_raytracing_tools *r)
{
	r->h_d_scene->photon_iteration = 1;
	gpu_errchk(hipHostMalloc(&r->rt_pixel_map, sizeof(t_color) *
		r->scene->res.y * r->scene->res.x));
	hipMemcpy(r->rt_pixel_map, r->d_pixel_map, sizeof(t_color) *
		r->scene->res.y * r->scene->res.x, hipMemcpyHostToHost);
	gpu_errchk(hipMalloc(&(r->h_d_scene->photon_list),
		sizeof(t_photon) * PHOTON_BOUNCE_MAX * r->scene->photons_per_pass));
	hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice);
}

void	copy_1(t_raytracing_tools *r)
{
	hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice);
}

void	copy_2(t_raytracing_tools *r)
{
	hipMemcpy(r->d_pixel_map, r->rt_pixel_map, sizeof(t_color) *
		r->scene->res.x * r->scene->res.y, hipMemcpyHostToHost);
}

void	free_map_and_list(t_raytracing_tools *r)
{
	hipHostFree(r->rt_pixel_map);
	hipFree(r->h_d_scene->photon_list);
}
