#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   c_min.cu                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jwalsh <jwalsh@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:24:09 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/20 17:20:53 by jwalsh           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Returns the min of col1 and col2.
*/

__host__ __device__
t_color	c_min(t_color col1, t_color col2)
{
	t_color	col;

	col.r = (col1.r < col2.r) ? col1.r : col2.r;
	col.g = (col1.g < col2.g) ? col1.g : col2.g;
	col.b = (col1.b < col2.b) ? col1.b : col2.b;
	return (col);
}
