#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   get_cone_intersection.cu                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/03/18 15:30:04 by talemari          #+#    #+#             */
/*   Updated: 2017/06/08 11:21:27 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../inc/rt.cuh"

__device__
void	get_finite_cone_intersection(t_ray *ray,
	t_object *object, t_intersection_tools *i);
__device__
bool	lower_than_min(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);
__device__
bool	higher_than_max(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray);

/*
** Checks for a ray-cone intersection.
*/

__device__
bool		get_cone_intersection(t_raytracing_tools *r, t_ray *ray,
			int index)
{
	t_intersection_tools i;

	i.d1 = tan(r->scene->objects[index].angle);
	i.v1 = v_sub(ray->origin, r->scene->objects[index].pos);
	i.q.x = v_dot(ray->dir, ray->dir) - (1.0 + i.d1 * i.d1) *
		powf(v_dot(ray->dir, r->scene->objects[index].dir), 2.0);
	i.q.y = 2 * (v_dot(ray->dir, i.v1) - (1.0 + i.d1 * i.d1) * v_dot(ray->dir,
		r->scene->objects[index].dir) *
		v_dot(i.v1, r->scene->objects[index].dir));
	i.q.z = v_dot(i.v1, i.v1) - (1.0 + i.d1 * i.d1) * powf(v_dot(i.v1,
		r->scene->objects[index].dir), 2.0);
	if (!solve_quadratic(i.q, &i.r1, &i.r2) || (i.r1 < 0 && i.r2 < 0))
		return (false);
	if (i.r2 < i.r1)
		ft_swapf(&i.r1, &i.r2);
	get_finite_cone_intersection(ray, &r->scene->objects[index], &i);
	(i.r1 < 0 || isnan(i.r1)) ? i.r1 = i.r2 : 0;
	if (i.r1 < 0 || isnan(i.r1))
		return (false);
	r->t > i.r1 ? ray->t = i.r1 : 0;
	(r->t > i.r1) ? ray->hit_obj = index : 0;
	(r->t > i.r1) ? ray->hit_type = T_CONE : 0;
	return (true);
}

__device__
void	get_finite_cone_intersection(t_ray *ray,
	t_object *obj, t_intersection_tools *i)
{
	bool	r1_too_low;
	bool	r1_too_high;
	bool	r2_too_low;
	bool	r2_too_high;

	r1_too_low = lower_than_min(i->r1, i, obj, ray) || i->r1 == -1;
	r1_too_high = higher_than_max(i->r1, i, obj, ray) || i->r1 == -1;
	r2_too_low = lower_than_min(i->r2, i, obj, ray) || i->r2 == -1;
	r2_too_high = higher_than_max(i->r2, i, obj, ray) || i->r2 == -1;
	if (r1_too_low || r1_too_high)
	{
		i->r1 = NAN;
		if (r2_too_low || r2_too_high)
			i->r2 = NAN;
	}
	else if (!r1_too_low && !r1_too_high)
		i->r2 = NAN;
}

__device__
bool	lower_than_min(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin,
			v_scale(ray->dir, r)), obj->pos)) < 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}

__device__
bool	higher_than_max(float r, t_intersection_tools *i, t_object *obj,
	t_ray *ray)
{
	if (r > 0)
	{
		if (v_dot(obj->dir, v_sub(v_add(ray->origin, v_scale(ray->dir, r)),
			v_add(obj->pos, v_scale(obj->dir, obj->height)))) > 0.0)
			return (true);
	}
	else
		return (true);
	return (false);
}

__device__
t_pt2		get_uv_cone(t_object *obj, t_ray *ray, t_vec3 *dim)
{
	t_pt2	coord;
	t_vec3	hit_center;

	hit_center = v_scale(ray->nhit, -1);
	coord.x = (0.5 + (atan2f(hit_center.z, hit_center.x) /
		(float)(2 * M_PI))) * dim->x;
	coord.y = (v_dist(obj->pos, ray->hit) /
		sqrtf(obj->height * obj->height + obj->rad * obj->rad)) * dim->y;
	coord.x %= (int)dim->x;
	coord.y %= (int)dim->y;
	return (coord);
}
