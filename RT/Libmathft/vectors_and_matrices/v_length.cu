#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   v_length.cu                                        :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: tgros <tgros@student.42.fr>                +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/02/21 13:24:51 by jwalsh            #+#    #+#             */
/*   Updated: 2017/04/26 11:17:01 by tgros            ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "libmathft.cuh"

/*
** Returns the length of the vector.
*/

__host__ __device__
float	v_length(t_vec3 v)
{
	return (sqrt(v_dot(v, v)));
}
