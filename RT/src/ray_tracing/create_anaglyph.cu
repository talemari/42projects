#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   create_anaglyph.cu                                 :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: talemari <talemari@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/05 09:52:05 by talemari          #+#    #+#             */
/*   Updated: 2017/06/09 10:17:06 by talemari         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "rt.cuh"
#include "../../inc/cuda_call.cuh"

__global__
void			get_look_at_position(t_scene *scene, t_vec3 *pos);
t_vec3			get_look_at(t_scene *scene);
void			update_camera(t_camera *camera);
__global__
void			create_anaglyph(t_color *left, t_color *right,
					t_scene *scene, t_tile tile);
__global__
void 			render_pixel(t_scene *scene, t_color *d_pixel_map,
					t_region *region_map, t_tile tile);

void				create_anaglyph_wrapper(t_raytracing_tools *r,
					dim3 block_size, dim3 grid_size, t_tile tile)
{
	t_vec3	original;
	
	original = r->scene->cameras->dir;
	r->scene->cameras->dir = v_norm(v_sub(get_look_at(r->d_scene),
		r->scene->cameras->pos));
	r->scene->cameras->pos.x += 0.08;
	update_camera(r->scene->cameras);
	r->scene->cameras->filter = F_RIGHT_CYAN;
	gpu_errchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras,
		sizeof(t_camera), hipMemcpyHostToDevice));
	gpu_errchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice)));
	render_pixel<<<grid_size, block_size>>>(r->d_scene, r->d_pixel_map_3d,
		r->d_region_map, tile);
	cuda_check_kernel_errors();
	r->scene->cameras->pos.x -= 0.08;
	r->scene->cameras->dir = original;
	update_camera(r->scene->cameras);
	r->scene->cameras->filter = F_LEFT_RED;
	gpu_errchk(hipMemcpy(r->h_d_scene->cameras, r->scene->cameras,
		sizeof(t_camera), hipMemcpyHostToDevice));
	gpu_errchk((hipMemcpy(r->d_scene, r->h_d_scene, sizeof(t_scene),
		hipMemcpyHostToDevice)));
	create_anaglyph<<<grid_size, block_size>>>(r->d_pixel_map,
		r->d_pixel_map_3d, r->d_scene, tile);
	cuda_check_kernel_errors();
}

t_vec3		get_look_at(t_scene *scene)
{
	t_vec3	h_look_at;
	t_vec3	*d_look_at;

	gpu_errchk(hipMalloc(&d_look_at, sizeof(t_vec3)));
	get_look_at_position<<<1, 1>>>(scene, d_look_at);
	cuda_check_kernel_errors();
	gpu_errchk(hipMemcpy(&h_look_at, d_look_at, sizeof(t_vec3),
		hipMemcpyDeviceToHost));
	return (h_look_at);
}

__global__
void			get_look_at_position(t_scene *scene, t_vec3 *pos)
{
	t_ray				cam_ray;
	t_raytracing_tools	r;
	t_dpt2				aa_i;
	int					i;

	r.pix.x = scene->res.x / 2.0;
	r.pix.y = scene->res.y / 2.0;
	r.scene = scene; 
    r.idx = scene->res.x * r.pix.y + r.pix.x;
	aa_i.x = 0.5;
	aa_i.y = 0.5;
	memset(&r.ior_list, 0, sizeof(float) * (MAX_RAY_DEPTH + 1));
	cam_ray = init_camera_ray(&r, aa_i);
	r.t = INFINITY; 
	i = -1;
	while (scene->objects[++i].type != T_INVALID_TOKEN)
	{
		if (intersects(&r, &cam_ray, i) && r.t > cam_ray.t)
			r.t = cam_ray.t;
	}
	if (r.t == INFINITY)
		*pos = v_new(NAN, NAN, NAN);
	else
		*pos = v_add(cam_ray.origin, v_scale(cam_ray.dir, r.t)); 
}

void			update_camera(t_camera *camera)
{
	t_vec3	forward;
	t_vec3	right;
	t_vec3	up;

	forward = v_norm(camera->dir);
	if (v_dot(forward, v_new(0, 1, 0)) > 0.9999 ||
		v_dot(forward, v_new(0, 1, 0)) < -0.9999)
		right = v_new(1, 0, 0);
	else
		right = v_norm(v_cross(v_new(0, 1, 0), forward));
	up = v_norm(v_cross(forward, right));
	m_new_identity(&camera->ctw);
	camera->ctw[0][0] = right.x;
	camera->ctw[0][1] = right.y;
	camera->ctw[0][2] = right.z;
	camera->ctw[1][0] = up.x;
	camera->ctw[1][1] = up.y;
	camera->ctw[1][2] = up.z;
	camera->ctw[2][0] = forward.x;
	camera->ctw[2][1] = forward.y;
	camera->ctw[2][2] = forward.z;
	camera->ctw[3][0] = camera->pos.x;
	camera->ctw[3][1] = camera->pos.y;
	camera->ctw[3][2] = camera->pos.z;
}

__global__
void			create_anaglyph(t_color *left, t_color *right, t_scene *scene,
					t_tile tile)
{
	int		idx;
	t_pt2	pixel;

	pixel.x = (tile.id.x * tile.size) + (blockDim.x * blockIdx.x) + threadIdx.x;
	pixel.y = (tile.id.y * tile.size) + (blockDim.y * blockIdx.y) + threadIdx.y;
  	idx = scene->res.x * pixel.y + pixel.x;

	if (pixel.x < scene->res.x && pixel.y < scene->res.y)
	{
		left[idx].g = right[idx].g;
		left[idx].b = right[idx].b;
	}
}
